#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "files.h"

#define SOFTENING 1e-9f

/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */

typedef struct { float x, y, z, vx, vy, vz; } Body;

/*
 * Calculate the gravitational impact of all bodies in the system
 * on all others.
 */
 
__global__
void bodyForce(Body *p, float dt, int n) {
  float Fx, Fy, Fz, dx, dy, dz, invDist, invDist3;
  int i, j, stride = blockDim.x * gridDim.x;
  
  for (i = threadIdx.x + blockIdx.x * blockDim.x; i < n; i += stride) {
    Fx = Fy = Fz = 0.0f;

    for (j = 0; j < n; j++) {
      dx = p[j].x - p[i].x;
      dy = p[j].y - p[i].y;
      dz = p[j].z - p[i].z;
      invDist = rsqrtf(dx*dx + dy*dy + dz*dz + SOFTENING);
      invDist3 = invDist * invDist * invDist;

      Fx += dx * invDist3;
      Fy += dy * invDist3;
      Fz += dz * invDist3;
    }

    p[i].vx += dt*Fx;
    p[i].vy += dt*Fy;
    p[i].vz += dt*Fz;
  }
  
}

__global__
void integratePosition(Body *p, float dt, int n){
    int i, stride = blockDim.x * gridDim.x;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < n; i += stride) {
        p[i].x += p[i].vx*dt;
        p[i].y += p[i].vy*dt;
        p[i].z += p[i].vz*dt;
    }
}


int main(const int argc, const char** argv) {

  // The assessment will test against both 2<11 and 2<15.
  // Feel free to pass the command line argument 15 when you generate ./nbody report files
  int nBodies = 2<<11;
  if (argc > 1) nBodies = 2<<atoi(argv[1]);
    
  int deviceId;
  hipGetDevice(&deviceId);  
  
  // The assessment will pass hidden initialized values to check for correctness.
  // You should not make changes to these files, or else the assessment will not work.
  const char * initialized_values;
  const char * solution_values;

  if (nBodies == 2<<11) {
    initialized_values = "09-nbody/files/initialized_4096";
    solution_values = "09-nbody/files/solution_4096";
  } else { // nBodies == 2<<15
    initialized_values = "09-nbody/files/initialized_65536";
    solution_values = "09-nbody/files/solution_65536";
  }

  if (argc > 2) initialized_values = argv[2];
  if (argc > 3) solution_values = argv[3];

  const float dt = 0.01f; // Time step
  const int nIters = 10;  // Simulation iterations

  int bytes = nBodies * sizeof(Body);
  float *buf;
  
  hipMallocManaged(&buf, bytes);
  Body *p = (Body*)buf;
  
  read_values_from_file(initialized_values, buf, bytes);
  hipMemPrefetchAsync(buf, bytes, deviceId);
  
  double totalTime = 0.0;

  /*
   * This simulation will run for 10 cycles of time, calculating gravitational
   * interaction amongst bodies, and adjusting their positions to reflect.
   */
  
  size_t n_threads = 512;
  size_t n_blocks = (nBodies + n_threads - 1)/n_threads;

  for (int iter = 0; iter < nIters; iter++) {
    StartTimer();

  /*
   * You will likely wish to refactor the work being done in `bodyForce`,
   * and potentially the work to integrate the positions.
   */

    bodyForce<<<n_blocks, n_threads>>>(p, dt, nBodies); // compute interbody forces
    hipDeviceSynchronize();
    
  /*
   * This position integration cannot occur until this round of `bodyForce` has completed.
   * Also, the next round of `bodyForce` cannot begin until the integration is complete.
   */

    integratePosition<<<n_blocks, n_threads>>>(p, dt, nBodies);
    hipDeviceSynchronize();
    
    const double tElapsed = GetTimer() / 1000.0;
    totalTime += tElapsed;
  }
  
  hipMemPrefetchAsync(buf, bytes, hipCpuDeviceId);
  
  double avgTime = totalTime / (double)(nIters);
  float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;
  write_values_to_file(solution_values, buf, bytes);

  // You will likely enjoy watching this value grow as you accelerate the application,
  // but beware that a failure to correctly synchronize the device might result in
  // unrealistically high values.
  printf("%0.3f Billion Interactions / second\n", billionsOfOpsPerSecond);

  free(buf);
}
