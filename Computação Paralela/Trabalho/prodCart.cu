
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <dirent.h>
#include <string.h>
#include <time.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>

typedef struct{
  unsigned long long v1;
  unsigned long long v2;
} Edge;

// Gets the position in an upper triangle adjacency matrix of the edge i-j
unsigned long long upper_triangle_position (unsigned long long i, unsigned long long j, unsigned long long vertices){
    // A beautiful example of branchless programming
    // If i>=j, switch j and i (false = 0, true = 1);
    return (i < j)*(i*vertices - (i*(i+1))/2 + j - i -1)
         + (i >=j)*(j*vertices - (j*(j+1))/2 + i - j -1);
}

// Returns the number of integers needed for the bitmap
unsigned long long bitmap_size_int (unsigned long long vertices){
    unsigned long long size = (vertices*(vertices-1))/2;
    return size/32+(size%32 != 0);
}

// Returns the number of bytes needed for the bitmap
unsigned long long bitmap_size_byte (unsigned long long vertices){
    return bitmap_size_int(vertices)*4;
}

// Inserts an edge i-j in a bitmap upper triangle adjacency matrix
void bitmap_write(unsigned int* bitmap, unsigned long long i, unsigned long long j, unsigned long long vertices){
    unsigned long long position = upper_triangle_position(i, j, vertices);
    unsigned int mask = 0x80000000 >> (position % 32);
    bitmap[position/32] = (mask | bitmap[position/32]);
    return;
}

// Gets the edge i-j in an upper triangle adjacency matrix
char bitmap_get(unsigned int bitmap[], unsigned long long i, unsigned long long j, unsigned long long vertices){
    unsigned long long position = upper_triangle_position(i, j, vertices);
    unsigned long long location = position/32;
    unsigned int mask = 0x80000000 >> (position % 32);
    char result = (mask & bitmap[location]) != 0;
    return result;
}

void generate_time_file_name(char time_file_name[], int dirIndex,
  char* vertices_string_G, char* edges_string_G, char* vertices_string_H, char* edges_string_H){
  
  strcpy(time_file_name, "./CUDAtime/");
  switch (dirIndex){
    case 0:
      strcat(time_file_name, "0.25x0.5-");
      break;
    case 1:
      strcat(time_file_name, "0.5x0.75-");
      break;
    case 2:
      strcat(time_file_name, "0.75x0.25-");
      break;
    default:
      printf("Time file name error\n");
      break;
  }

  strcat(time_file_name, vertices_string_G);
  strcat(time_file_name, ".");
  strcat(time_file_name, edges_string_G);
  strcat(time_file_name, "x");
  strcat(time_file_name, vertices_string_H);
  strcat(time_file_name, ".");
  strcat(time_file_name, edges_string_H);
  strcat(time_file_name, ".time");
}

void generate_result_file_name(char result_file_name[], int dirIndex,
  char* vertices_string_G, char* edges_string_G, char* vertices_string_H, char* edges_string_H){
  
  strcpy(result_file_name, "./CUDAresult/");
  switch (dirIndex){
    case 0:
      strcat(result_file_name, "0.25x0.5-");
      break;
    case 1:
      strcat(result_file_name, "0.5x0.75-");
      break;
    case 2:
      strcat(result_file_name, "0.75x0.25-");
      break;
    default:
      printf("Result file name error\n");
      break;
  }

  strcat(result_file_name, vertices_string_G);
  strcat(result_file_name, ".");
  strcat(result_file_name, edges_string_G);
  strcat(result_file_name, "x");
  strcat(result_file_name, vertices_string_H);
  strcat(result_file_name, ".");
  strcat(result_file_name, edges_string_H);
  strcat(result_file_name, ".result");
}

void write_time_file (char time_file_name[], clock_t delta_time){
  FILE *time_file;
  time_file = fopen(time_file_name, "w");
  fprintf(time_file, "%ld\n",delta_time);
  fclose(time_file);
}

void write_result_file (char result_file_name[], unsigned int P[], unsigned long long vertices_P){
  FILE *result_file;
  unsigned long long i, j;
  result_file = fopen(result_file_name, "w");
  for (i = 0; i < vertices_P; i++){
		for (j = i+1; j < vertices_P; j++)
			if (bitmap_get(P, i, j, vertices_P))
        fprintf (result_file, "%llu-%llu\n", i,j);
	}
  fclose(result_file);
}

// Creating edge list from a edge list file
Edge* edge_array_from_file(char file_name[], unsigned int edges){
  unsigned long long i, v1, v2;
  FILE *file;
  Edge* edge_list;

  // Opening graph file
  file = fopen(file_name, "r");
  if (!file){
    printf("Couldn't open file %s\n", file_name);
    return 0;
  }

  // Constructing H edge list
  edge_list = (Edge*) malloc(edges*sizeof(Edge));
  if (!edge_list)
    return NULL;
  
  for (i = 0; fscanf(file,"%llu %llu", &v1, &v2) != EOF; i++){
    edge_list[i].v1 = v1;
    edge_list[i].v2 = v2;
  }
  fclose(file);

  return edge_list;
}

// Print the edge array passing the number of edges
void printEdgeArray (Edge* edge_list, unsigned long long edges){
	int i;
	for (i = 0; i < edges; i++)
		printf("%llu-%llu\n", edge_list[i].v1, edge_list[i].v2);
	return;
}

// Print the Bitmap of an Upper Triangle as an edge list
void print_bitmap_upper (unsigned int P[], unsigned long long vertices_P){
	unsigned long long i, j;
	for (i = 0; i < vertices_P; i++){
		for (j = i+1; j < vertices_P; j++)
			if (bitmap_get(P, i, j, vertices_P))
        printf ("%llu-%llu\n", i,j);
	}
	return;
}

// Calculates part of the cartesian product P = G x H iterating only edges of G
// P is a bitmap of the adjancy matrix upper triangle
__global__
void cartProdBitmapG (Edge* G, unsigned int* P, unsigned long long edges_G, unsigned long long vertices_H, unsigned long long vertices_P){
  unsigned long long m, position;
  unsigned long long v1P, v2P;
  unsigned int mask;

  unsigned long long idx = (unsigned long long)blockIdx.x * (unsigned long long)blockDim.x + (unsigned long long)threadIdx.x;
  unsigned long long stride = (unsigned long long)gridDim.x * (unsigned long long)blockDim.x;
  for (; idx < edges_G; idx += stride){
    //printf("G: %llu-%llu\n", G[idx].v1, G[idx].v2);
    for (m = 0; m < vertices_H; m++){
      v1P = G[idx].v1*vertices_H + m;
      v2P = G[idx].v2*vertices_H + m;
      
      position = (v1P < v2P)*(v1P*vertices_P - (v1P*(v1P+1))/2 + v2P - v1P -1)
        + (v1P >= v2P)*(v2P*vertices_P - (v2P*(v2P+1))/2 + v1P - v2P -1);
      mask = 0x80000000 >> (position % 32);
      
      // To evict running condition, use atomic OR
      atomicOr(&P[position/32], mask);
		}
  }
	return;
}

// Calculates part of the cartesian product P = G x H iterating only edges of H
// P is a bitmap of the adjancy matrix upper triangle
__global__
void cartProdBitmapH (Edge* H, unsigned int* P, unsigned long long edges_H, unsigned long long vertices_G, unsigned long long vertices_P){
  unsigned long long m, position;
  unsigned long long v1P, v2P;
  unsigned int mask;

  unsigned long long idx = (unsigned long long)blockIdx.x * (unsigned long long)blockDim.x + (unsigned long long)threadIdx.x;
  unsigned long long stride = (unsigned long long)gridDim.x * (unsigned long long)blockDim.x;
  
  for (; idx < edges_H; idx += stride){
  //printf("H: %llu-%llu\n", H[idx].v1, H[idx].v2);
    for (m = 0; m < vertices_G; m++){
      v1P = m*vertices_G + H[idx].v1;
      v2P = m*vertices_G + H[idx].v2; 

      position = (v1P < v2P)*(v1P*vertices_P - (v1P*(v1P+1))/2 + v2P - v1P -1)
        + (v1P >= v2P)*(v2P*vertices_P - (v2P*(v2P+1))/2 + v1P - v2P -1);
      mask = 0x80000000 >> (position % 32);
      
      // To evict running condition, use atomic OR
      atomicOr(&P[position/32], mask);
		}
  }



	return;
}

int main(){
  unsigned long long vertices_G, vertices_H, vertices_P, edges_G, edges_H;
  int dirIndex;
  char file_name_G[100], file_name_H[100], time_file_name[100], result_file_name[100];
  char *vertices_string_G, *edges_string_G, *vertices_string_H, *edges_string_H;
  const char *directories_G[3] = {"./0.25/\0","./0.5/\0","./0.75/\0"};
  const char *directories_H[3] = {"./0.5/\0","./0.75/\0","./0.25/\0"};
  Edge *edge_array_G_RAM, *edge_array_H_RAM, *edge_array_G_VRAM, *edge_array_H_VRAM;
  unsigned int *bitmap_P_VRAM;
  unsigned int *bitmap_P_RAM;
  DIR *d_G, *d_H;
  clock_t delta_time;
  struct stat st = {0};
  struct dirent *dir_G;
  struct dirent *dir_H;
  size_t len_P_bytes;
  size_t len_P_ints;
  size_t threadsPerBlock;
  size_t numberOfBlocks;
  int deviceId;
  int numberOfSMs;
  hipError_t error;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  // Setting CUDA kernel execution parameters
  threadsPerBlock = 256;
  numberOfBlocks = 1 * numberOfSMs;

  // Create directory CUDAtime/ if it doesn't exist 
  if (stat("CUDAtime", &st) == -1) {
      mkdir("CUDAtime", 0700);
  }

  // Create directory CUDAresult/ if it doesn't exist 
  if (stat("CUDAresult", &st) == -1) {
      mkdir("CUDAresult", 0700);
  }

  // For each of the three combination of graph densities
  for (dirIndex=0; dirIndex < 3; dirIndex++){ 

    // Opening directory
    d_G = opendir(directories_G[dirIndex]);
    d_H = opendir(directories_H[dirIndex]);
    if (!d_G || !d_H){
      printf ("Couldn't open directory\n");
      return 0;
    }

    printf ("Starting %s x %s batch\n\n", directories_G[dirIndex], directories_H[dirIndex]);

    // Reading files of the directory
    while (1) {
      
      dir_G = readdir(d_G);
      dir_H = readdir(d_H);
      
      while (dir_G && dir_G->d_name[0] == '.') // Skipping . and ..
        dir_G = readdir(d_G);
      while (dir_H && dir_H->d_name[0] == '.')
        dir_H = readdir(d_H);
      
      if (!dir_G || !dir_H) // If one of the directories reached the end
        break;

      // Mounting the filename
      strcpy(file_name_G, directories_G[dirIndex]);
      strcat(file_name_G, dir_G->d_name);

      strcpy(file_name_H, directories_H[dirIndex]);
      strcat(file_name_H, dir_H->d_name);
      //printf ("Filenames: %s x %s\n", file_name_G, file_name_H);

      // Getting the number of vertices and edges in the filename
      vertices_string_G = strtok(dir_G->d_name, ".");
      edges_string_G = strtok(NULL, ".");
      vertices_G = atoi(vertices_string_G);
      edges_G = atoi(edges_string_G);

      vertices_string_H = strtok(dir_H->d_name, ".");
      edges_string_H = strtok(NULL, ".");
      vertices_H = atoi(vertices_string_H);
      edges_H = atoi(edges_string_H);

      edge_array_G_RAM = edge_array_from_file(file_name_G, edges_G);
      edge_array_H_RAM = edge_array_from_file(file_name_H, edges_H);

      //printEdgeList(edge_array_G_RAM, edges_G);
      //printEdgeList(edge_array_H_RAM, edges_H);
      
      // Initializing host structure
      vertices_P = vertices_G*vertices_H;
      len_P_bytes = bitmap_size_byte(vertices_P);
      len_P_ints = bitmap_size_int(vertices_P);
      bitmap_P_RAM = (unsigned int *) calloc (len_P_ints, sizeof(unsigned int));

      //print_bitmap_upper(bitmap_P, vertices_P);

      // Starting to calculate the cartesian product
      printf("Calculating %s.%s x %s.%s... ",vertices_string_G, edges_string_G, vertices_string_H, edges_string_H);
      fflush(stdout);
      delta_time = clock();

      // ------------------------------------------------------------------
      // Initializing device structures
      error = hipMalloc(&bitmap_P_VRAM, len_P_bytes);
      if(error != hipSuccess)
        printf("CUDA MALLOC BITMAP IN DEVICE error: %s\n", hipGetErrorString(error));
      error = hipMalloc(&edge_array_G_VRAM, edges_G*sizeof(Edge));
      if(error != hipSuccess)
        printf("CUDA MALLOC EDGES G IN DEVICE error: %s\n", hipGetErrorString(error));
      error = hipMalloc(&edge_array_H_VRAM, edges_H*sizeof(Edge));
      if(error != hipSuccess)
        printf("CUDA MALLOC EDGES H IN DEVICE error: %s\n", hipGetErrorString(error));
      error = hipMemset(bitmap_P_VRAM, 0, len_P_bytes);
      if(error != hipSuccess)
        printf("CUDA MEM SET BITMAP IN DEVICE error: %s\n", hipGetErrorString(error));
      error = hipMemcpy(edge_array_G_VRAM, edge_array_G_RAM, edges_G*sizeof(Edge), hipMemcpyHostToDevice); // Synchronous
      if(error != hipSuccess)
        printf("CUDA MEM CPY G TO DEVICE error: %s\n", hipGetErrorString(error));
      error = hipMemcpy(edge_array_H_VRAM, edge_array_H_RAM, edges_H*sizeof(Edge), hipMemcpyHostToDevice); // Synchronous
      if(error != hipSuccess)
        printf("CUDA MEM CPY H TO DEVICE error: %s\n", hipGetErrorString(error));

      // Calculating the cartesian product
      cartProdBitmapG <<<numberOfBlocks, threadsPerBlock>>> (edge_array_G_VRAM, bitmap_P_VRAM, edges_G, vertices_H, vertices_P);
      cartProdBitmapH <<<numberOfBlocks, threadsPerBlock>>> (edge_array_H_VRAM, bitmap_P_VRAM, edges_H, vertices_G, vertices_P);
      hipDeviceSynchronize();
      
      error = hipGetLastError();
      if(error != hipSuccess)
        printf("CUDA PROCESSING error: %s\n", hipGetErrorString(error));

      // Copying the result from device to host
      error = hipMemcpy(bitmap_P_RAM, bitmap_P_VRAM, len_P_bytes, hipMemcpyDeviceToHost); // Synchronous
      if(error != hipSuccess)
        printf("CUDA MEM CPY BITMAP TO HOST error: %s\n", hipGetErrorString(error));
      // ------------------------------------------------------------------

      // Calculating passed time
      delta_time = clock() - delta_time;
      printf("Finished!\n");
      //print_bitmap_upper(bitmap_P, vertices_P);

      /*
      // Saving result as a file (consumes a LOT of time)
      generate_result_file_name(result_file_name, dirIndex,
        vertices_string_G, edges_string_G, vertices_string_H, edges_string_H);
      write_result_file(result_file_name, bitmap_P_RAM, vertices_P);
      */

      // Freeing result and edge lists
      hipFree(bitmap_P_VRAM);
      hipFree(edge_array_G_VRAM);
      hipFree(edge_array_H_VRAM);
      free(bitmap_P_RAM);
      free(edge_array_G_RAM);
      free(edge_array_H_RAM);

      // Saving the time as a file
      generate_time_file_name(time_file_name, dirIndex,
        vertices_string_G, edges_string_G, vertices_string_H, edges_string_H);
      printf("Time saved as %s\n\n", time_file_name);
      write_time_file (time_file_name, delta_time);
    }
    closedir(d_G);
    closedir(d_H);

    printf ("Ended %s x %s batch!\n\n", directories_G[dirIndex], directories_H[dirIndex]);
  }
	return 0;
}