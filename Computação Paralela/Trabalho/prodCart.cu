
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <dirent.h>
#include <string.h>
#include <sys/time.h> 
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>

typedef struct{
  unsigned long long v1;
  unsigned long long v2;
} Edge;

// Gets the position in an upper triangle adjacency matrix of the edge i-j
unsigned long long upper_triangle_position (unsigned long long i, unsigned long long j, unsigned long long vertices){
    // A beautiful example of branchless programming
    // If i>=j, switch j and i (false = 0, true = 1);
    return (i < j)*(i*vertices - (i*(i+1))/2 + j - i -1)
         + (i >=j)*(j*vertices - (j*(j+1))/2 + i - j -1);
}

// Returns the number of integers needed for the bitmap
unsigned long long bitmap_size_int (unsigned long long vertices){
    unsigned long long size = (vertices*(vertices-1))/2;
    return size/32+(size%32 != 0);
}

// Returns the number of bytes needed for the bitmap
unsigned long long bitmap_size_byte (unsigned long long vertices){
    return bitmap_size_int(vertices)*4;
}

// Inserts an edge i-j in a bitmap upper triangle adjacency matrix
void bitmap_write(unsigned int* bitmap, unsigned long long i, unsigned long long j, unsigned long long vertices){
    unsigned long long position = upper_triangle_position(i, j, vertices);
    unsigned int mask = 0x80000000 >> (position % 32);
    bitmap[position/32] = (mask | bitmap[position/32]);
    return;
}

// Gets the edge i-j in an upper triangle adjacency matrix
char bitmap_get(unsigned int bitmap[], unsigned long long i, unsigned long long j, unsigned long long vertices){
    unsigned long long position = upper_triangle_position(i, j, vertices);
    unsigned long long location = position/32;
    unsigned int mask = 0x80000000 >> (position % 32);
    char result = (mask & bitmap[location]) != 0;
    return result;
}

// Generates the file name for saving the time
// TYPE: 0 = host-to-device, 1 = processing, 2 = device-to-host
void generate_time_file_name(char time_file_name[], int type, int dirIndex,
  char* vertices_string_G, char* edges_string_G, char* vertices_string_H, char* edges_string_H){
  
  switch (type){
    case 0:
      strcpy(time_file_name, "./CUDAtime/host_to_device-");
      break;
    case 1:
      strcpy(time_file_name, "./CUDAtime/processing-");
      break;
    case 2:
      strcpy(time_file_name, "./CUDAtime/device_to_host-");
      break;
    default:
      printf("Time file name error\n");
      return;
  }

  switch (dirIndex){
    case 0:
      strcat(time_file_name, "0.25x0.5-");
      break;
    case 1:
      strcat(time_file_name, "0.5x0.75-");
      break;
    case 2:
      strcat(time_file_name, "0.75x0.25-");
      break;
    default:
      printf("Time file name error\n");
      return;
  }

  strcat(time_file_name, vertices_string_G);
  strcat(time_file_name, ".");
  strcat(time_file_name, edges_string_G);
  strcat(time_file_name, "x");
  strcat(time_file_name, vertices_string_H);
  strcat(time_file_name, ".");
  strcat(time_file_name, edges_string_H);
  strcat(time_file_name, ".time");
}

void generate_result_file_name(char result_file_name[], int dirIndex,
  char* vertices_string_G, char* edges_string_G, char* vertices_string_H, char* edges_string_H){
  
  strcpy(result_file_name, "./CUDAresult/");
  switch (dirIndex){
    case 0:
      strcat(result_file_name, "0.25x0.5-");
      break;
    case 1:
      strcat(result_file_name, "0.5x0.75-");
      break;
    case 2:
      strcat(result_file_name, "0.75x0.25-");
      break;
    default:
      printf("Result file name error\n");
      return;
  }

  strcat(result_file_name, vertices_string_G);
  strcat(result_file_name, ".");
  strcat(result_file_name, edges_string_G);
  strcat(result_file_name, "x");
  strcat(result_file_name, vertices_string_H);
  strcat(result_file_name, ".");
  strcat(result_file_name, edges_string_H);
  strcat(result_file_name, ".result");
}

void write_time_file (char time_file_name[], double time){
  FILE *time_file;
  time_file = fopen(time_file_name, "w");
  fprintf(time_file, "%lf\n",time);
  fclose(time_file);
}

void write_result_file (char result_file_name[], unsigned int P[], unsigned long long vertices_P){
  FILE *result_file;
  unsigned long long i, j;
  result_file = fopen(result_file_name, "w");
  for (i = 0; i < vertices_P; i++){
		for (j = i+1; j < vertices_P; j++)
			if (bitmap_get(P, i, j, vertices_P))
        fprintf (result_file, "%llu-%llu\n", i,j);
	}
  fclose(result_file);
}

// Creating edge list from a edge list file
Edge* edge_array_from_file(char file_name[], unsigned int edges){
  unsigned long long i, v1, v2;
  FILE *file;
  Edge* edge_list;

  // Opening graph file
  file = fopen(file_name, "r");
  if (!file){
    printf("Couldn't open file %s\n", file_name);
    return 0;
  }

  // Constructing H edge list
  edge_list = (Edge*) malloc(edges*sizeof(Edge));
  if (!edge_list)
    return NULL;
  
  for (i = 0; fscanf(file,"%llu %llu", &v1, &v2) != EOF; i++){
    edge_list[i].v1 = v1;
    edge_list[i].v2 = v2;
  }
  fclose(file);

  return edge_list;
}

// Print the edge array passing the number of edges
void printEdgeArray (Edge* edge_list, unsigned long long edges){
	int i;
	for (i = 0; i < edges; i++)
		printf("%llu-%llu\n", edge_list[i].v1, edge_list[i].v2);
	return;
}

// Print the Bitmap of an Upper Triangle as an edge list
void print_bitmap_upper (unsigned int P[], unsigned long long vertices_P){
	unsigned long long i, j;
	for (i = 0; i < vertices_P; i++){
		for (j = i+1; j < vertices_P; j++)
			if (bitmap_get(P, i, j, vertices_P))
        printf ("%llu-%llu\n", i,j);
	}
	return;
}

// Calculates part of the cartesian product P = G x H iterating only edges of G
// P is a bitmap of the adjancy matrix upper triangle
__global__
void cartProdBitmapG (Edge* G, unsigned int* P, unsigned long long edges_G, unsigned long long vertices_H, unsigned long long vertices_P){
  unsigned long long m, position;
  unsigned long long v1P, v2P;
  unsigned int mask;

  unsigned long long idx = (unsigned long long)blockIdx.x * (unsigned long long)blockDim.x + (unsigned long long)threadIdx.x;
  unsigned long long stride = (unsigned long long)gridDim.x * (unsigned long long)blockDim.x;
  for (; idx < edges_G; idx += stride){
    //printf("G: %llu-%llu\n", G[idx].v1, G[idx].v2);
    for (m = 0; m < vertices_H; m++){
      v1P = G[idx].v1*vertices_H + m;
      v2P = G[idx].v2*vertices_H + m;
      
      position = (v1P < v2P)*(v1P*vertices_P - (v1P*(v1P+1))/2 + v2P - v1P -1)
        + (v1P >= v2P)*(v2P*vertices_P - (v2P*(v2P+1))/2 + v1P - v2P -1);
      mask = 0x80000000 >> (position % 32);
      
      // To evict running condition, use atomic OR
      atomicOr(&P[position/32], mask);
		}
  }
	return;
}

// Calculates part of the cartesian product P = G x H iterating only edges of H
// P is a bitmap of the adjancy matrix upper triangle
__global__
void cartProdBitmapH (Edge* H, unsigned int* P, unsigned long long edges_H, unsigned long long vertices_G, unsigned long long vertices_H, unsigned long long vertices_P){
  unsigned long long m, position;
  unsigned long long v1P, v2P;
  unsigned int mask;

  unsigned long long idx = (unsigned long long)blockIdx.x * (unsigned long long)blockDim.x + (unsigned long long)threadIdx.x;
  unsigned long long stride = (unsigned long long)gridDim.x * (unsigned long long)blockDim.x;
  
  for (; idx < edges_H; idx += stride){
  //printf("H: %llu-%llu\n", H[idx].v1, H[idx].v2);
    for (m = 0; m < vertices_G; m++){
      v1P = m*vertices_H + H[idx].v1;
      v2P = m*vertices_H + H[idx].v2; 

      position = (v1P < v2P)*(v1P*vertices_P - (v1P*(v1P+1))/2 + v2P - v1P -1)
        + (v1P >= v2P)*(v2P*vertices_P - (v2P*(v2P+1))/2 + v1P - v2P -1);
      mask = 0x80000000 >> (position % 32);
      
      // To evict running condition, use atomic OR
      atomicOr(&P[position/32], mask);
		}
  }
	return;
}

int main(int argc, char** argv){
  unsigned long long vertices_G, vertices_H, vertices_P, edges_G, edges_H;
  int dirIndex;
  char file_name_G[100], file_name_H[100], time_file_name[100], result_file_name[100];
  char *vertices_string_G, *edges_string_G, *vertices_string_H, *edges_string_H;
  const char *directories_G[3] = {"./0.25/\0","./0.5/\0","./0.75/\0"};
  const char *directories_H[3] = {"./0.5/\0","./0.75/\0","./0.25/\0"};
  Edge *edge_array_G_RAM, *edge_array_H_RAM, *edge_array_G_VRAM, *edge_array_H_VRAM;
  unsigned int *bitmap_P_VRAM;
  unsigned int *bitmap_P_RAM;
  DIR *d_G, *d_H;
  struct timeval start_time, end_time;
  double host_to_device_time, device_to_host_time, processing_time;
  struct stat st = {0};
  struct dirent *dir_G;
  struct dirent *dir_H;
  size_t len_P_bytes;
  size_t len_P_ints;
  size_t threadsPerBlock;
  size_t numberOfBlocks;
  int deviceId;
  int numberOfSMs;
  int arg_n_threads;
  int arg_n_blocks;
  hipError_t error;

  // Dealing with arguments
  if (argc != 3){
    printf("Must receive 2 arguments: <n_threads_per_block> <n_blocks_per_SM>");
    return 0;
  }
  arg_n_threads = atoi(argv[1]);
  arg_n_blocks = atoi(argv[2]);

  // Getting device info
  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  // Setting CUDA kernel execution parameters
  threadsPerBlock = arg_n_threads;
  numberOfBlocks = arg_n_blocks * numberOfSMs;

  // Create directory CUDAtime/ if it doesn't exist 
  if (stat("CUDAtime", &st) == -1) {
      mkdir("CUDAtime", 0700);
  }

  // Create directory CUDAresult/ if it doesn't exist 
  if (stat("CUDAresult", &st) == -1) {
      mkdir("CUDAresult", 0700);
  }

  // For each of the three combination of graph densities
  for (dirIndex=0; dirIndex < 3; dirIndex++){ 

    // Opening directory
    d_G = opendir(directories_G[dirIndex]);
    d_H = opendir(directories_H[dirIndex]);
    if (!d_G || !d_H){
      printf ("Couldn't open directory\n");
      return 0;
    }

    printf ("Starting %s x %s batch\n\n", directories_G[dirIndex], directories_H[dirIndex]);

    // Reading files of the directory
    while (1) {
      
      dir_G = readdir(d_G);
      dir_H = readdir(d_H);
      
      while (dir_G && dir_G->d_name[0] == '.') // Skipping . and ..
        dir_G = readdir(d_G);
      while (dir_H && dir_H->d_name[0] == '.')
        dir_H = readdir(d_H);
      
      if (!dir_G || !dir_H) // If one of the directories reached the end
        break;

      // Mounting the filename
      strcpy(file_name_G, directories_G[dirIndex]);
      strcat(file_name_G, dir_G->d_name);

      strcpy(file_name_H, directories_H[dirIndex]);
      strcat(file_name_H, dir_H->d_name);
      //printf ("Filenames: %s x %s\n", file_name_G, file_name_H);

      // Getting the number of vertices and edges in the filename
      vertices_string_G = strtok(dir_G->d_name, ".");
      edges_string_G = strtok(NULL, ".");
      vertices_G = atoi(vertices_string_G);
      edges_G = atoi(edges_string_G);

      vertices_string_H = strtok(dir_H->d_name, ".");
      edges_string_H = strtok(NULL, ".");
      vertices_H = atoi(vertices_string_H);
      edges_H = atoi(edges_string_H);

      edge_array_G_RAM = edge_array_from_file(file_name_G, edges_G);
      edge_array_H_RAM = edge_array_from_file(file_name_H, edges_H);
      
      // Initializing host structure
      vertices_P = vertices_G*vertices_H;
      len_P_bytes = bitmap_size_byte(vertices_P);
      len_P_ints = bitmap_size_int(vertices_P);
      bitmap_P_RAM = (unsigned int *) calloc (len_P_ints, sizeof(unsigned int));

      // Setting up the result bitmap
      error = hipMalloc(&bitmap_P_VRAM, len_P_bytes);
      if(error != hipSuccess)
        printf("CUDA MALLOC BITMAP IN DEVICE error: %s\n", hipGetErrorString(error));
      error = hipMemset(bitmap_P_VRAM, 0, len_P_bytes);
      if(error != hipSuccess)
        printf("CUDA MEM SET BITMAP IN DEVICE error: %s\n", hipGetErrorString(error));

      // Starting host-to-device memory transfer timer
      gettimeofday(&start_time, NULL);
      
      // Setting up G
      error = hipMalloc(&edge_array_G_VRAM, edges_G*sizeof(Edge));
      if(error != hipSuccess)
        printf("CUDA MALLOC EDGES G IN DEVICE error: %s\n", hipGetErrorString(error));
      error = hipMemcpy(edge_array_G_VRAM, edge_array_G_RAM, edges_G*sizeof(Edge), hipMemcpyHostToDevice); // Synchronous
      if(error != hipSuccess)
        printf("CUDA MEM CPY G TO DEVICE error: %s\n", hipGetErrorString(error));

      // Setting up H
      error = hipMalloc(&edge_array_H_VRAM, edges_H*sizeof(Edge));
      if(error != hipSuccess)
        printf("CUDA MALLOC EDGES H IN DEVICE error: %s\n", hipGetErrorString(error));
      error = hipMemcpy(edge_array_H_VRAM, edge_array_H_RAM, edges_H*sizeof(Edge), hipMemcpyHostToDevice); // Synchronous
      if(error != hipSuccess)
        printf("CUDA MEM CPY H TO DEVICE error: %s\n", hipGetErrorString(error));
      
      // Saving host-to-device memory transfer timer
      gettimeofday(&end_time, NULL);
      host_to_device_time = (end_time.tv_sec - start_time.tv_sec) * 1000.0 + (end_time.tv_usec - start_time.tv_usec)/1000.0;

      // Starting to calculate the cartesian product
      printf("Calculating %s.%s x %s.%s... ",vertices_string_G, edges_string_G, vertices_string_H, edges_string_H);
      fflush(stdout);
      
      // ------------------------------------------------------------------ 
      // Starting processing timer
      gettimeofday(&start_time, NULL);

      // Calculating the cartesian product
      cartProdBitmapG <<<numberOfBlocks, threadsPerBlock>>> (edge_array_G_VRAM, bitmap_P_VRAM, edges_G, vertices_H, vertices_P);
      cartProdBitmapH <<<numberOfBlocks, threadsPerBlock>>> (edge_array_H_VRAM, bitmap_P_VRAM, edges_H, vertices_G, vertices_H, vertices_P);
      hipDeviceSynchronize();
      
      /*
      error = cudaGetLastError();
      if(error != cudaSuccess)
        printf("CUDA PROCESSING error: %s\n", cudaGetErrorString(error));
      */

      // Saving processing timer
      gettimeofday(&end_time, NULL);
      processing_time = (end_time.tv_sec - start_time.tv_sec) * 1000.0 + (end_time.tv_usec - start_time.tv_usec)/1000.0;
      // ------------------------------------------------------------------
      printf("Finished!\n");
      
      // Starting device-to-host memory transfer timer
      gettimeofday(&start_time, NULL);

      // Copying the result from device to host
      error = hipMemcpy(bitmap_P_RAM, bitmap_P_VRAM, len_P_bytes, hipMemcpyDeviceToHost); // Synchronous
      if(error != hipSuccess)
        printf("CUDA MEM CPY BITMAP TO HOST error: %s\n", hipGetErrorString(error));

      // Saving device-to-host memory transfer timer
      gettimeofday(&end_time, NULL);
      device_to_host_time = (end_time.tv_sec - start_time.tv_sec) * 1000.0 + (end_time.tv_usec - start_time.tv_usec)/1000.0;
      
      /*
      // Saving result as a file (consumes a LOT of time)
      generate_result_file_name(result_file_name, dirIndex,
        vertices_string_G, edges_string_G, vertices_string_H, edges_string_H);
      write_result_file(result_file_name, bitmap_P_RAM, vertices_P);
      */

      // Freeing result and edge lists
      hipFree(bitmap_P_VRAM);
      hipFree(edge_array_G_VRAM);
      hipFree(edge_array_H_VRAM);
      free(bitmap_P_RAM);
      free(edge_array_G_RAM);
      free(edge_array_H_RAM);

      printf("Host to device memory transfer time (ms): %lf\n", host_to_device_time);
      printf("Processing time (ms): %lf\n", processing_time);
      printf("Device to host memory transfer time (ms): %lf\n", device_to_host_time);

      // Saving times as files
      generate_time_file_name(time_file_name, 0, dirIndex,
        vertices_string_G, edges_string_G, vertices_string_H, edges_string_H);
      write_time_file (time_file_name, host_to_device_time);
      printf("Host to device memory transfer time saved as %s\n", time_file_name);

      generate_time_file_name(time_file_name, 1, dirIndex,
        vertices_string_G, edges_string_G, vertices_string_H, edges_string_H);
      write_time_file (time_file_name, processing_time);
      printf("Processing time saved as %s\n", time_file_name);

      generate_time_file_name(time_file_name, 2, dirIndex,
        vertices_string_G, edges_string_G, vertices_string_H, edges_string_H);
      write_time_file (time_file_name, device_to_host_time);
      printf("Device to host memory transfer time saved as %s\n\n", time_file_name);

    }
    closedir(d_G);
    closedir(d_H);

    printf ("Ended %s x %s batch!\n\n", directories_G[dirIndex], directories_H[dirIndex]);
  }
	return 0;
}