
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int gridStride = blockDim.x * gridDim.x;
  for(; i < N; i+=gridStride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);
  
  size_t n_threads_per_block = 1024;
  size_t n_blocks = (n_threads_per_block + N - 1)/n_threads_per_block;
  
  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);
    
  addVectorsInto<<<n_blocks, n_threads_per_block>>>(c, a, b, N);
  hipDeviceSynchronize();
  
  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
