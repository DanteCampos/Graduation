
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

__global__ void loop(int N)
{
   int i = blockDim.x * blockIdx.x + threadIdx.x;
   if (i < N)
       printf("This is iteration number %d\n", i);
}

int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, be sure to use more than 1 block in
   * the execution configuration.
   */

  int N = 100;
  int n_threads_per_block = 16;
  int n_blocks = (N + n_threads_per_block - 1)/n_threads_per_block;
  loop<<<n_blocks, n_threads_per_block>>>(N);
  hipDeviceSynchronize();
}
