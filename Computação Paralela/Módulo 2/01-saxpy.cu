#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 2048 * 2048 // Number of elements in each vector

/*
 * Optimize this already-accelerated codebase. Work iteratively,
 * and use nsys to support your work.
 *
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 20us.
 *
 * Some bugs have been placed in this codebase for your edification.
 */

__global__
void initMem(float *a, int x){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if ( tid < N )
        a[tid] = x;
}

__global__ void saxpy(float * a, float * b, float * c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if ( tid < N )
        c[tid] = 2 * a[tid] + b[tid];
}

int main()
{
    float *a, *b, *c;

    int size = N * sizeof (float); // The total number of bytes per vector
    
    int deviceId;
    hipGetDevice(&deviceId);
    
    int threads_per_block = 256;
    int number_of_blocks = N / threads_per_block;
    
    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);
    
    hipMemPrefetchAsync(a, size, deviceId);
    hipMemPrefetchAsync(b, size, deviceId);
    hipMemPrefetchAsync(c, size, deviceId);
    
    // Initialize memory
    initMem<<< number_of_blocks, threads_per_block >>> (a, 2);
    initMem<<< number_of_blocks, threads_per_block >>> (b, 1);
    initMem<<< number_of_blocks, threads_per_block >>> (c, 0);
    
    hipDeviceSynchronize();
    
    saxpy <<< number_of_blocks, threads_per_block >>> ( a, b, c ); 
    hipDeviceSynchronize();
    
    hipMemPrefetchAsync(c, size, deviceId);
    
    // Print out the first and last 5 values of c for a quality check
    for( int i = 0; i < 5; ++i )
        printf("c[%d] = %f, ", i, c[i]);
    printf ("\n");
    for( int i = N-5; i < N; ++i )
        printf("c[%d] = %f, ", i, c[i]);
    printf ("\n");

    hipFree( a ); hipFree( b ); hipFree( c );
}
